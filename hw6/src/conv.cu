#include <conv.h>
#include <utils.h>

size_t conv::num_blocks(size_t image_size, size_t num_thread_per_block)
{
    return (image_size + num_thread_per_block - 1) / num_thread_per_block;
}

void conv::CudaDeleter::operator()(void * p) const
{
     hipFree(p);
}

float * conv::cuda_allocate(size_t size_in_bytes)
{
    float * buffer = nullptr;
    hipMalloc(&buffer, size_in_bytes);
    hipMemset(buffer, 0, size_in_bytes);
    return buffer;
}

void conv::conv2d(const float * input_image, std::vector<float>& output_image, size_t width, size_t height, size_t num_c, const std::vector<float> & kernel_buffer, float normalization, size_t kernel_size)
{
    const int THREAD_PER_BLOCK = 16;

    int num_width_blocks = conv::num_blocks(width, THREAD_PER_BLOCK);
    int num_height_blocks = conv::num_blocks(height, THREAD_PER_BLOCK);

    size_t num_bytes = width * height * num_c * sizeof(float);

    std::unique_ptr<float[], CudaDeleter> device_image_buffer(cuda_allocate(num_bytes));
    std::unique_ptr<float[], CudaDeleter> device_image_out_buffer(cuda_allocate(num_bytes));
    std::unique_ptr<float[], CudaDeleter> device_kernel_buffer(cuda_allocate(num_bytes));

    auto error = hipMemcpy(device_image_buffer.get(), input_image, num_bytes, hipMemcpyHostToDevice);

    utils::check_and_print_error(error);

    error = hipMemcpy(device_kernel_buffer.get(), kernel_buffer.data(), kernel_buffer.size() * sizeof(float), hipMemcpyHostToDevice);

    utils::check_and_print_error(error);

    gpu::conv2d_kernel<<<dim3(num_width_blocks, num_height_blocks), dim3(THREAD_PER_BLOCK, THREAD_PER_BLOCK, num_c)>>>(device_image_buffer.get(), device_image_out_buffer.get(), width, height, num_c, device_kernel_buffer.get(), normalization, kernel_size);

    error = hipPeekAtLastError();

    utils::check_and_print_error(error);

    error = hipMemcpy(output_image.data(), device_image_out_buffer.get(), num_bytes, hipMemcpyDeviceToHost);

    utils::check_and_print_error(error);
}
